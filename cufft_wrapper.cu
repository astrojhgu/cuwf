#include "cufft_wrapper.h"
#include <cstdio>
#include <cstdlib>
#include <hipfft/hipfft.h>


struct FftResources
{
    int npt;
    int nbatch;
    struct fcomplex *d_input;
    struct fcomplex *d_output;
    hipfftHandle plan;
};

#define CHECK_CUDA(call)                                                         \
    do {                                                                         \
        hipError_t err = (call);                                                \
        if (err != hipSuccess) {                                                \
            fprintf(stderr, "CUDA Error at %s:%d: %s\n",                         \
                    __FILE__, __LINE__, hipGetErrorString(err));               \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    } while (0)

// cuFFT 错误检查宏
#define CHECK_CUFFT(call)                                                        \
    do {                                                                         \
        hipfftResult err = (call);                                                \
        if (err != HIPFFT_SUCCESS) {                                              \
            fprintf(stderr, "cuFFT Error at %s:%d: %s\n",                        \
                    __FILE__, __LINE__, cufftGetErrorString(err));              \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    } while (0)


const char* cufftGetErrorString(hipfftResult error) {
    switch (error) {
        case HIPFFT_SUCCESS: return "HIPFFT_SUCCESS";
        case HIPFFT_INVALID_PLAN: return "HIPFFT_INVALID_PLAN";
        case HIPFFT_ALLOC_FAILED: return "HIPFFT_ALLOC_FAILED";
        case HIPFFT_INVALID_TYPE: return "HIPFFT_INVALID_TYPE";
        case HIPFFT_INVALID_VALUE: return "HIPFFT_INVALID_VALUE";
        case HIPFFT_INTERNAL_ERROR: return "HIPFFT_INTERNAL_ERROR";
        case HIPFFT_EXEC_FAILED: return "HIPFFT_EXEC_FAILED";
        case HIPFFT_SETUP_FAILED: return "HIPFFT_SETUP_FAILED";
        case HIPFFT_INVALID_SIZE: return "HIPFFT_INVALID_SIZE";
        case HIPFFT_UNALIGNED_DATA: return "HIPFFT_UNALIGNED_DATA";
        default: return "Unknown CUFFT error";
    }
}

extern "C" FftResources* init_resources(int npt, int nbatch) {
    FftResources *resources=new FftResources;
    resources->npt = npt;
    resources->nbatch = nbatch;
    CHECK_CUFFT(hipfftPlan1d(&resources->plan, npt, HIPFFT_C2C, nbatch));
    CHECK_CUDA(hipMalloc((void**)&resources->d_input, sizeof(fcomplex) * npt*nbatch));
    CHECK_CUDA(hipMalloc((void**)&resources->d_output, sizeof(fcomplex) * npt*nbatch));
    return resources;
}

extern "C" void destroy_resources(FftResources *resources) {
    CHECK_CUFFT(hipfftDestroy(resources->plan));
    CHECK_CUDA(hipFree(resources->d_input));
    CHECK_CUDA(hipFree(resources->d_output));
    delete resources;
}

extern "C" void fft_execute(FftResources *resources, fcomplex *h_input, fcomplex *h_output) {
    CHECK_CUDA(hipMemcpy(resources->d_input, h_input, sizeof(fcomplex) * resources->npt*resources->nbatch, hipMemcpyHostToDevice));
    CHECK_CUFFT(hipfftExecC2C(resources->plan, (hipfftComplex*)resources->d_input, (hipfftComplex*)resources->d_output, HIPFFT_FORWARD));
    CHECK_CUDA(hipMemcpy(h_output, resources->d_output, sizeof(fcomplex) * resources->npt*resources->nbatch, hipMemcpyDeviceToHost));
}

extern "C" void fft_execute_inverse(FftResources *resources, fcomplex *h_input, fcomplex *h_output) {
    CHECK_CUDA(hipMemcpy(resources->d_input, h_input, sizeof(hipfftComplex) * resources->npt*resources->nbatch, hipMemcpyHostToDevice));
    CHECK_CUFFT(hipfftExecC2C(resources->plan, (hipfftComplex*)resources->d_input, (hipfftComplex*)resources->d_output, HIPFFT_BACKWARD));
    CHECK_CUDA(hipMemcpy(h_output, resources->d_output, sizeof(hipfftComplex) * resources->npt*resources->nbatch, hipMemcpyDeviceToHost));
}
